#include "hip/hip_runtime.h"
#include "VBW_sc.hh"
#include "LFuncGpu.hh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void calc(double *x, double *mix_saxs_1d, double alpha_zero, float *fitSaxsMix, int total) {

	float deltamix, smix;

	int i= blockIdx.x * blockDim.x + threadIdx.x;
	int j= blockIdx.y * blockDim.y + threadIdx.y;

	if (i>=total || j>=total) return;

	smix = mix_saxs_1d[i*total+j];
        deltamix = (i!=j) ? -x[i]*x[j] : x[i]*(alpha_zero - x[i]);
	//Most likely atomic add will be needed
	//fitSaxsMix += 1.0;
        fitSaxsMix[i] =smix * deltamix;

}

void checkCuda(hipError_t result) {
  	if (result != hipSuccess) {
    		std::cerr<<"CUDA Runtime Error: "<<hipGetErrorString(result);
		std::exit(0);
  	}
}

void LFuncLoopGpu (block * x, double * mix_saxs_1d, double alpha_zero, double fit_saxs_mix, int total) {

	float *h_f;
	float *d_h; 
	double *d_x, *d_m;
	h_f = ( float * ) malloc( sizeof( float ) * total ) ;
	checkCuda ( hipMalloc( (void **)&d_h, sizeof( float ) * total));
	checkCuda ( hipMalloc( (void **)&d_x, sizeof( double ) * total));
	checkCuda ( hipMalloc( (void **)&d_m, sizeof( double ) * total * total));	
    	checkCuda ( hipMemcpy( d_h, h_f, sizeof( float ) * total, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy( d_x, x->alphas, sizeof( double ) * total, hipMemcpyHostToDevice) );
	checkCuda ( hipMemcpy( d_m, mix_saxs_1d, sizeof( double ) * total * total, hipMemcpyHostToDevice) );
	//int gpu_processor = 1;
	//checkCuda ( hipSetDevice( gpu_processor ) );
	int gpuBlockSize = 32;
	dim3 GpuBlock( gpuBlockSize,gpuBlockSize );
	dim3 GpuGrid (  (total+gpuBlockSize-1)/gpuBlockSize,  (total+gpuBlockSize-1)/gpuBlockSize  );
	calc<<<GpuGrid,GpuBlock>>>(d_x, d_m, alpha_zero, d_h, total);
	hipDeviceSynchronize();
	checkCuda( hipMemcpy(h_f, d_h, sizeof( float ) * total, hipMemcpyDeviceToHost) );
	//std::cout<<"Copy from Device"<<std::endl;
	for(int i=0; i< total; i++) fit_saxs_mix+=h_f[i];
	free(h_f);
	hipFree( d_h );
	hipFree( d_x );
	hipFree( d_m );
	//hipDeviceSynchronize();
}
